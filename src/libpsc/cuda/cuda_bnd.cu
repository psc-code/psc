#include "hip/hip_runtime.h"

#include "cuda_iface.h"
#include "cuda_iface_bnd.h"
#include "cuda_bits.h"

#include "psc.h"
#include "psc_fields_cuda.h"
#include "fields.hxx"

#include "cuda_bnd.cuh"

template <typename real_t>
__global__ static void k_scatter_add(const real_t* buf, const uint* map,
                                     real_t* flds, unsigned int size)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < size) {
    atomicAdd(&flds[map[i]], buf[i]);
  }
}

template <typename real_t>
void ScatterAdd::operator()(const thrust::host_vector<uint>& map,
                            const thrust::host_vector<real_t>& buf,
                            thrust::host_vector<real_t>& h_flds)
{
  auto p = buf.begin();
  for (auto cur : map) {
    h_flds[cur] += *p++;
  }
}

template <typename real_t>
void ScatterAdd::operator()(const psc::device_vector<uint>& map,
                            const psc::device_vector<real_t>& buf,
                            thrust::device_ptr<real_t> d_flds)
{
  if (buf.empty())
    return;

  const int THREADS_PER_BLOCK = 256;
  dim3 dimGrid((buf.size() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  k_scatter_add<<<dimGrid, THREADS_PER_BLOCK>>>(
    buf.data().get(), map.data().get(), d_flds.get(), buf.size());
  cuda_sync_if_enabled();
}

// ======================================================================
// Scatter

template <typename real_t>
__global__ static void k_scatter(const real_t* buf, const uint* map,
                                 real_t* flds, unsigned int size)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < size) {
    flds[map[i]] = buf[i];
  }
}

template <typename real_t>
void Scatter::operator()(const thrust::host_vector<uint>& map,
                         const thrust::host_vector<real_t>& buf,
                         thrust::host_vector<real_t>& h_flds)
{
  thrust::scatter(buf.begin(), buf.end(), map.begin(), h_flds.begin());
}

template <typename real_t>
void Scatter::operator()(const psc::device_vector<uint>& map,
                         const psc::device_vector<real_t>& buf,
                         thrust::device_ptr<real_t> d_flds)
{
#if 1
  thrust::scatter(buf.begin(), buf.end(), map.begin(), d_flds);
#else
  if (buf.empty())
    return;

  const int THREADS_PER_BLOCK = 256;
  dim3 dimGrid((buf.size() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  k_scatter<<<dimGrid, THREADS_PER_BLOCK>>>(buf.data().get(), map.data().get(),
                                            d_flds.get(), buf.size());
  cuda_sync_if_enabled();
#endif
}

// ======================================================================

template void ScatterAdd::operator()(const thrust::host_vector<uint>& map,
                                     const thrust::host_vector<float>& buf,
                                     thrust::host_vector<float>& h_flds);
template void ScatterAdd::operator()(const psc::device_vector<uint>& map,
                                     const psc::device_vector<float>& buf,
                                     thrust::device_ptr<float> d_flds);

template void Scatter::operator()(const thrust::host_vector<uint>& map,
                                  const thrust::host_vector<float>& buf,
                                  thrust::host_vector<float>& h_flds);
template void Scatter::operator()(const psc::device_vector<uint>& map,
                                  const psc::device_vector<float>& buf,
                                  thrust::device_ptr<float> d_flds);

// ======================================================================

#define BLOCKSIZE_X 1
#define BLOCKSIZE_Y 4
#define BLOCKSIZE_Z 4

#define SW (2) // FIXME

template <bool lo, bool hi, typename E>
__global__ static void conducting_wall_H_y(E gt, Int3 ib)
{
  int iz = blockIdx.x * blockDim.x + threadIdx.x - SW;

  auto _d_flds = make_Fields3d<dim_xyz>(gt, ib);

  if (iz >= gt.shape(2) - SW)
    return;

  int my = gt.shape(1) - 2 * SW;

  if (lo) {
    _d_flds(HY, 0, -1, iz) = _d_flds(HY, 0, 1, iz);
    _d_flds(HX, 0, -1, iz) = -_d_flds(HX, 0, 0, iz);
    _d_flds(HZ, 0, -1, iz) = -_d_flds(HZ, 0, 0, iz);
  }

  if (hi) {
    _d_flds(HY, 0, my + 1, iz) = _d_flds(HY, 0, my - 1, iz);
    _d_flds(HX, 0, my, iz) = -_d_flds(HX, 0, my - 1, iz);
    _d_flds(HZ, 0, my, iz) = -_d_flds(HZ, 0, my - 1, iz);
  }
}

template <bool lo, bool hi, typename E>
__global__ static void conducting_wall_E_y(E gt, Int3 ib)
{
  int iz = blockIdx.x * blockDim.x + threadIdx.x - SW;

  auto _d_flds = make_Fields3d<dim_xyz>(gt, ib);

  if (iz >= gt.shape(2) - SW)
    return;

  int my = gt.shape(1) - 2 * SW;

  if (lo) {
    _d_flds(EX, 0, 0, iz) = 0.;
    _d_flds(EX, 0, -1, iz) = _d_flds(EX, 0, 1, iz);
    _d_flds(EY, 0, -1, iz) = -_d_flds(EY, 0, 0, iz);
    _d_flds(EZ, 0, 0, iz) = 0.;
    _d_flds(EZ, 0, -1, iz) = _d_flds(EZ, 0, 1, iz);
  }

  if (hi) {
    _d_flds(EX, 0, my, iz) = 0.;
    _d_flds(EX, 0, my + 1, iz) = _d_flds(EX, 0, my - 1, iz);
    _d_flds(EY, 0, my, iz) = -_d_flds(EY, 0, my - 1, iz);
    _d_flds(EZ, 0, my, iz) = 0.;
    _d_flds(EZ, 0, my + 1, iz) = _d_flds(EZ, 0, my - 1, iz);
  }
}

template <bool lo, bool hi, typename E>
__global__ static void conducting_wall_J_y(E gt, Int3 ib)
{
  int iz = blockIdx.x * blockDim.x + threadIdx.x - SW;

  auto _d_flds = make_Fields3d<dim_xyz>(gt, ib);

  if (iz >= gt.shape(2) - SW)
    return;

  int my = gt.shape(1) - 2 * SW;

  if (lo) {
    _d_flds(JYI, 0, 0, iz) -= _d_flds(JYI, 0, -1, iz);
    _d_flds(JYI, 0, -1, iz) = 0.;
    _d_flds(JXI, 0, 1, iz) += _d_flds(JXI, 0, -1, iz);
    _d_flds(JXI, 0, -1, iz) = 0.;
    _d_flds(JZI, 0, 1, iz) += _d_flds(JZI, 0, -1, iz);
    _d_flds(JZI, 0, -1, iz) = 0.;
  }

  if (hi) {
    _d_flds(JYI, 0, my - 1, iz) -= _d_flds(JYI, 0, my, iz);
    _d_flds(JYI, 0, my, iz) = 0.;
    _d_flds(JXI, 0, my - 1, iz) += _d_flds(JXI, 0, my + 1, iz);
    _d_flds(JXI, 0, my + 1, iz) = 0.;
    _d_flds(JZI, 0, my - 1, iz) += _d_flds(JZI, 0, my + 1, iz);
    _d_flds(JZI, 0, my + 1, iz) = 0.;
  }
}

template <bool lo, bool hi>
static void cuda_conducting_wall_H_y(MfieldsCuda& mflds, int p)
{
  int dimGrid = (mflds.gt().shape(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z;
  conducting_wall_H_y<lo, hi>
    <<<dimGrid, BLOCKSIZE_Z>>>(view_patch(mflds.gt(), p), -mflds.ibn());
  cuda_sync_if_enabled();
}

template <bool lo, bool hi>
static void cuda_conducting_wall_E_y(MfieldsCuda& mflds, int p)
{
  int dimGrid = (mflds.gt().shape(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z;
  conducting_wall_E_y<lo, hi>
    <<<dimGrid, BLOCKSIZE_Z>>>(view_patch(mflds.gt(), p), -mflds.ibn());
  cuda_sync_if_enabled();
}

template <bool lo, bool hi>
static void cuda_conducting_wall_J_y(MfieldsCuda& mflds, int p)
{
  int dimGrid = (mflds.gt().shape(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z;
  conducting_wall_J_y<lo, hi>
    <<<dimGrid, BLOCKSIZE_Z>>>(view_patch(mflds.gt(), p), -mflds.ibn());
  cuda_sync_if_enabled();
}

void cuda_conducting_wall_H_lo_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_H_y<true, false>(mflds, p);
}

void cuda_conducting_wall_H_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_H_y<false, true>(mflds, p);
}

void cuda_conducting_wall_H_lo_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_H_y<true, true>(mflds, p);
}

void cuda_conducting_wall_E_lo_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_E_y<true, false>(mflds, p);
}

void cuda_conducting_wall_E_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_E_y<false, true>(mflds, p);
}

void cuda_conducting_wall_E_lo_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_E_y<true, true>(mflds, p);
}

void cuda_conducting_wall_J_lo_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_J_y<true, false>(mflds, p);
}

void cuda_conducting_wall_J_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_J_y<false, true>(mflds, p);
}

void cuda_conducting_wall_J_lo_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_J_y<true, true>(mflds, p);
}
